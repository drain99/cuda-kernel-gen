#include "hip/hip_runtime.h"
#include "kernels.cuh"
__global__ 
void kernel__0(int* in_1, float* in_2, float* out) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < 25)
		out[idx] = (in_1[idx]-in_2[idx]);
}
__global__ 
void kernel__1(int* in_1_1, int* in_1_2, int* in_2_1_1, int* in_2_1_2, float* in_2_2, float* out) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < 25)
		out[idx] = ((in_1_1[idx]+in_1_2[idx])*((in_2_1_1[idx]+in_2_1_2[idx])-in_2_2[idx]));
}
__global__ 
void kernel__2(int* in_1_1_1_1, int* in_1_1_1_2, int* in_1_1_2_1_1, int* in_1_1_2_1_2, float* in_1_1_2_2, int* in_1_2, int* in_2, float* out) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < 25)
		out[idx] = ((((in_1_1_1_1[idx]+in_1_1_1_2[idx])*((in_1_1_2_1_1[idx]+in_1_1_2_1_2[idx])-in_1_1_2_2[idx]))+in_1_2[idx])-in_2[idx]);
}
